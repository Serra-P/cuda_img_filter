#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloWorldKernel() {
    printf("Hello, World! from GPU\n");
}

int main() {
    std::cout << "Hello, World! from CPU" << std::endl;

    // Lancer le kernel CUDA
    helloWorldKernel<<<1, 1>>>();

    // Synchroniser pour s'assurer que le kernel a terminé
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    return 0;
}
